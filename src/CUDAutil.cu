#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <cstdlib>

#include "stdcuda.h"

// ------------------------------------------------------------------------------
int findAndSetCUDADevice(int nDev,int verbose) {
	
int devCount;
CC(hipGetDeviceCount(&devCount));

size_t mem_tot;
size_t mem_free;
hipDeviceProp_t devProp;
for (int thisDev=0;thisDev<devCount;thisDev++) {
  CC(hipGetDeviceProperties(&devProp,thisDev));
  CC(hipMemGetInfo(&mem_free, &mem_tot));
  if (verbose) {
    printf("Device %d found %s with CC=%d%d\n",thisDev,devProp.name,devProp.major,devProp.minor);
    printf("Total memory %zu and free %zu\n",mem_tot,mem_free);
  }
}

if (nDev>=devCount) { printf("GPU devices does not exist"); return (1); }
else {
  CC(hipGetDeviceProperties(&devProp,nDev));
  CC(hipSetDevice(nDev));
  CC(hipDeviceReset());
  if (verbose) printf("Using %s\n",devProp.name);
}
return (0);
}
// ------------------------------------------------------------------------------
